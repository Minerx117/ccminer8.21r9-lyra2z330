extern "C" {
#include "sph/sph_blake.h"
#include "sph/sph_groestl.h"
#include "sph/sph_skein.h"
#include "sph/sph_keccak.h"
#include "lyra2/Lyra2.h"
}

#include "miner.h"
#include "cuda_helper.h"
#include <hip/hip_runtime_api.h>
static _ALIGN(64) uint64_t *d_hash[MAX_GPUS];
static THREAD uint32_t *foundNonce;


extern void blake256_cpu_hash_80(const int thr_id, const uint32_t threads, const uint32_t startNonce, uint64_t *Hash);
extern void blake256_cpu_setBlock_80(int thr_id, uint32_t *pdata);
extern void keccak256_cpu_hash_32(int thr_id, uint32_t threads, uint32_t startNonce, uint64_t *d_outputHash);
extern void keccak256_cpu_init(int thr_id, uint32_t threads);
extern void skein256_cpu_hash_32(int thr_id, uint32_t threads, uint32_t startNonce, uint64_t *d_outputHash);
extern void skein256_cpu_init(int thr_id, uint32_t threads);

extern void lyra2_cpu_hash_32(int thr_id, uint32_t threads, uint32_t startNonce, uint64_t *d_outputHash);
extern void lyra2_cpu_hash_32_multi(int thr_id, uint32_t threads, uint32_t startNonce, uint64_t *d_outputHash);

extern void groestl256_setTarget(int thr_id, const void *ptarget);
extern void lyra2_cpu_init(int thr_id, uint32_t threads);
extern void lyra2_cpu_init_multi(int thr_id, uint32_t threads, uint64_t *hash, uint64_t* hash2);
extern void groestl256_cpu_hash_32(int thr_id, uint32_t threads, uint32_t startNounce, uint64_t *d_outputHash, uint32_t *resultnonces);
extern void groestl256_cpu_init(int thr_id, uint32_t threads);

extern "C" void lyra2_hash(void *state, const void *input)
{
	sph_blake256_context     ctx_blake;
	sph_keccak256_context    ctx_keccak;
	sph_skein256_context     ctx_skein;
	sph_groestl256_context   ctx_groestl;

	uint32_t hashA[8], hashB[8];

	sph_blake256_init(&ctx_blake);
	sph_blake256(&ctx_blake, input, 80);
	sph_blake256_close(&ctx_blake, hashA);

	sph_keccak256_init(&ctx_keccak);
	sph_keccak256(&ctx_keccak, hashA, 32);
	sph_keccak256_close(&ctx_keccak, hashB);

	LYRA2_old(hashA, 32, hashB, 32, hashB, 32, 1, 8, 8, TRUE);
	sph_skein256_init(&ctx_skein);
	sph_skein256(&ctx_skein, hashA, 32);
	sph_skein256_close(&ctx_skein, hashB);

	sph_groestl256_init(&ctx_groestl);
	sph_groestl256(&ctx_groestl, hashB, 32);
	sph_groestl256_close(&ctx_groestl, hashA);

	memcpy(state, hashA, 32);
}

static volatile bool init[MAX_GPUS] = { false };

extern int scanhash_lyra2(int thr_id, uint32_t *pdata,
	uint32_t *ptarget, uint32_t max_nonce,
	uint32_t *hashes_done)
{
	const uint32_t first_nonce = pdata[19];
	unsigned int intensity = (device_sm[device_map[thr_id]] > 500) ? 256 * 256 * 4 : 256 * 256 * 4 ;
    intensity = (device_sm[device_map[thr_id]] == 500) ? 256 * 256 * 2 : intensity;
	uint32_t throughput = device_intensity(device_map[thr_id], __func__, intensity); // 18=256*256*4;
	

	if (opt_benchmark)
		ptarget[7] = 0x00ff;

	
	if(!init[thr_id])
	{ 
		CUDA_SAFE_CALL(hipSetDevice(device_map[thr_id]));
		hipSetDeviceFlags(hipDeviceScheduleBlockingSync);
		hipDeviceSetCacheConfig(hipFuncCachePreferL1);
		CUDA_SAFE_CALL(hipStreamCreate(&gpustream[thr_id]));
		CUDA_SAFE_CALL(hipProfilerStop());
		CUDA_SAFE_CALL(hipHostMalloc(&foundNonce, 2 * 4));
		CUDA_SAFE_CALL(hipMalloc(&d_hash[thr_id], 8 * sizeof(uint32_t) * throughput));
		keccak256_cpu_init(thr_id, throughput);
		skein256_cpu_init(thr_id, throughput);
		groestl256_cpu_init(thr_id, throughput);
		lyra2_cpu_init(thr_id, throughput);

		init[thr_id] = true; 
	}
	else
		CUDA_SAFE_CALL(hipProfilerStart());

	uint32_t endiandata[20];
	for (int k=0; k < 20; k++)
		be32enc(&endiandata[k], pdata[k]);

	blake256_cpu_setBlock_80(thr_id, pdata);
	groestl256_setTarget(thr_id, ptarget);
	do {
		blake256_cpu_hash_80(thr_id, throughput, pdata[19], d_hash[thr_id]);
		keccak256_cpu_hash_32(thr_id, throughput, pdata[19], d_hash[thr_id]);
		lyra2_cpu_hash_32(thr_id, throughput, pdata[19], d_hash[thr_id]);
		skein256_cpu_hash_32(thr_id, throughput, pdata[19], d_hash[thr_id]);
		groestl256_cpu_hash_32(thr_id, throughput, pdata[19], d_hash[thr_id], foundNonce);
		CUDA_SAFE_CALL(hipGetLastError());
		if(stop_mining) {mining_has_stopped[thr_id] = true; hipStreamDestroy(gpustream[thr_id]); pthread_exit(nullptr);}
		if(foundNonce[0] != 0)
		{
			const uint32_t Htarg = ptarget[7];
			uint32_t vhash64[8];
			be32enc(&endiandata[19], foundNonce[0]);
			lyra2_hash(vhash64, endiandata);
			if (vhash64[7] <= Htarg && fulltest(vhash64, ptarget))
			{
				int res = 1;
				// check if there was some other ones...
				*hashes_done = pdata[19] - first_nonce + throughput;
				if (foundNonce[1] != 0)
				{
					be32enc(&endiandata[19], foundNonce[1]);
					lyra2_hash(vhash64, endiandata);

					if (vhash64[7] <= Htarg && fulltest(vhash64, ptarget))
					{
						pdata[21] = foundNonce[1];
						res++;
						if (opt_benchmark)  applog(LOG_INFO, "GPU #%d: Found second nounce %08x", device_map[thr_id], foundNonce[1]);
					}
					else
					{
						if (vhash64[7] != Htarg) // don't show message if it is equal but fails fulltest
							applog(LOG_WARNING, "GPU #%d: result %08x does not validate on CPU!", device_map[thr_id], foundNonce[1]);
					}
				}
				pdata[19] = foundNonce[0];
				if (opt_benchmark) applog(LOG_INFO, "GPU #%d: Found nounce %08x", device_map[thr_id], foundNonce[0]);
				return res;
			}
			else
			{
				if (vhash64[7] != Htarg) // don't show message if it is equal but fails fulltest
					applog(LOG_WARNING, "GPU #%d: result %08x does not validate on CPU!", device_map[thr_id], foundNonce[0]);
			}
		}

		pdata[19] += throughput;

	} while (!work_restart[thr_id].restart && ((uint64_t)max_nonce > ((uint64_t)(pdata[19]) + (uint64_t)throughput)));

	*hashes_done = pdata[19] - first_nonce + 1;
	return 0;
}
